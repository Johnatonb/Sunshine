#include "hip/hip_runtime.h"
﻿#include <fstream>
#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include "vec3.cuh"
#include "ray.cuh"
#include "hittable.cuh"
#include "hittable_list.cuh"
#include "sphere.cuh"
#include "camera.cuh"
#include "material.cuh"
#include "moving_sphere.cuh"
#include "bvh.cuh"
#include "txture.cuh"
#include "perlin.cuh"
#include "aarect.cuh"
#include "box.cuh"
#include <vector>
#include <cmath>
#include <stdio.h>


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hittable** world, const vec3& background, hiprandState* local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation(1.0f, 1.0f, 1.0f);
    for (int i = 0; i < 5; i++) {
        hit_record rec;
        if (!(*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            return background;
        }
        else {
            ray scattered;
            vec3 attenuation;
            vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            if (!rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                return emitted * cur_attenuation;
            }
            cur_attenuation *= attenuation;
            cur_ray = scattered;
        }
    }
    return vec3(0.0f, 0.0f, 0.0f);
    
}

__global__ void render(vec3* data, int max_x, int max_y, int samples, camera** cam, hittable** world,  hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int index = j * max_x + i;
    hiprandState local_rand_state = rand_state[index];
    vec3 col(0.0f, 0.0f, 0.0f);
    for (int s = 0; s < samples; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, vec3(0.0f,0.0f,0.0f), & local_rand_state);
    }
    col /= float(samples);
    for (int i = 0; i < 3; i++) {
        if (col[i] > 1) { col[i] = 1; }
    }
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    data[index] = col;
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int index = j * max_x + i;
    hiprand_init(1984 + index, 0, 0, &rand_state[index]);
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera, hiprandState* rand_state, vec3 lookfrom, vec3 lookat, vec3 vup, float vfov, float aspect_ratio) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;

        d_list[0] = new yz_rect(0.0f, 555.0f, 0.0f, 555.0f, 555.0f, new lambertian(vec3(0.12f, 0.45f, 0.15f)));
        d_list[1] = new yz_rect(0.0f, 555.0f, 0.0f, 555.0f, 0.0f, new lambertian(vec3(0.65f, 0.05f, 0.05f)));
        d_list[2] = new xz_rect(213.0f, 343.0f, 227.0f, 332.0f, 554.0f, new diffuse_light(vec3(15.0f, 15.0f, 15.0f)));
        d_list[3] = new xz_rect(0.0f, 555.0f, 0.0f, 555.0f, 0.0f, new lambertian(vec3(0.73f, 0.73f, 0.73f)));
        d_list[4] = new xz_rect(0.0f, 555.0f, 0.0f, 555.0f, 555.0f, new lambertian(vec3(0.73f, 0.73f, 0.73f)));
        d_list[5] = new xy_rect(0.0f, 555.0f, 0.0f, 555.0f, 555.0f, new lambertian(vec3(0.73f, 0.73f, 0.73f)));

        d_list[6] = new sphere(vec3(255.0f, 255.0f, 255.0f), 50.0f, new lambertian(vec3(0.73f, 0.73f, 0.73f)));

        *rand_state = local_rand_state;
        *d_world = new hittable_list(d_list, 7);
        //*d_world = new bvh_node(hittable_list(d_list, 6), 6, 0.0f, 1.0f, &local_rand_state);
        *d_camera = new camera(lookfrom, lookat, vup, vfov, aspect_ratio, 0.1f, 10.f, 0.0f, 1.0f);
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete ((sphere*)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_camera;
    delete *d_world;
}

int main() {
    const auto aspect_ratio = 1.0f;
    const int width = 512;
    const int height = static_cast<int> (width / aspect_ratio);
    int num_pixels = width * height;
    int samples = 1000;

    vec3* image;
    checkCudaErrors(hipMallocManaged((void**)&image, num_pixels * sizeof(vec3)));

    hittable** d_list;
    int num_hittables = 30;
    hittable** d_world;
    hiprandState* d_rand_state;
    camera** d_camera;

    checkCudaErrors(hipMalloc((void**)&d_list, num_hittables * sizeof(hittable*)));
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));

    float threads = 8.0f;
    dim3 BlocksPerGrid = dim3(ceil(width / threads), ceil(height / threads));
    dim3 threadsPerBlock = dim3(threads, threads);
    render_init <<<BlocksPerGrid, threadsPerBlock>>> (width, height, d_rand_state);
    create_world <<<1,1>>> (d_list, d_world, d_camera, d_rand_state, vec3(278,278,-800), vec3(278, 278, 0), vec3(0, 1, 0), 40, aspect_ratio);
    render <<<BlocksPerGrid, threadsPerBlock >>> (image, width, height, samples, d_camera, d_world, d_rand_state);
    hipMemcpy(image, image, num_pixels, hipMemcpyHostToDevice);

    std::cout << "P3\n" << width << ' ' << height << "\n255\n";
    for (int j = height - 1; j >= 0; --j) {
        for (int i = 0; i < width; ++i) {

            size_t pixel_index = j * width + i;
            int ir = static_cast<int>(255.999 * image[pixel_index].r());
            int ig = static_cast<int>(255.999 * image[pixel_index].g());
            int ib = static_cast<int>(255.999 * image[pixel_index].b());

            std::cout << ir << ' ' << ig << ' ' << ib << '\n';
        }
    }

    free_world <<<1,1>>> (d_list, d_world, d_camera);
}